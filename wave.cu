/**********************************************************************
 * DESCRIPTION:
 *   Serial Concurrent Wave Equation - C Version
 *   This program implements the concurrent wave equation
 *********************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265
#define Num 32

void check_param(void);
void init_line(void);
void update (void);
void printfinal (void);

int nsteps,                 	/* number of time steps */
    tpoints, 	     		/* total points along string */
    rcode;                  	/* generic return code */
float  values[MAXPOINTS+2], 	/* values at time t */
       oldval[MAXPOINTS+2], 	/* values at time (t-dt) */
       newval[MAXPOINTS+2]; 	/* values at time (t+dt) */


/**********************************************************************
 *	Checks input values from parameters
 *********************************************************************/
void check_param(void)
{
   char tchar[20];

   /* check number of points, number of iterations */
   while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
      printf("Enter number of points along vibrating string [%d-%d]: "
           ,MINPOINTS, MAXPOINTS);
      scanf("%s", tchar);
      tpoints = atoi(tchar);
      if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
         printf("Invalid. Please enter value between %d and %d\n", 
                 MINPOINTS, MAXPOINTS);
   }
   while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
      printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
      scanf("%s", tchar);
      nsteps = atoi(tchar);
      if ((nsteps < 1) || (nsteps > MAXSTEPS))
         printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
   }

   printf("Using points = %d, steps = %d\n", tpoints, nsteps);

}

/**********************************************************************
 *     Initialize points on line
 *********************************************************************/
/*void init_line(void)
{
   int i, j;
   float x, fac, k, tmp;

   // Calculate initial values based on sine curve 
   fac = 2.0 * PI;
   k = 0.0; 
   tmp = tpoints - 1;
   for (j = 1; j <= tpoints; j++) {
      x = k/tmp;
      values[j] = sin (fac * x);
      k = k + 1.0;
   } 

   //Initialize old values array
   for (i = 1; i <= tpoints; i++) 
      oldval[i] = values[i];
}*/

/**********************************************************************
 *      Calculate new values using wave equation
 *********************************************************************/
void do_math(int i)
{
   float dtime, c, dx, tau, sqtau;

   dtime = 0.3;
   c = 1.0;
   dx = 1.0;
   tau = (c * dtime / dx);
   sqtau = tau * tau;
   newval[i] = (2.0 * values[i]) - oldval[i] + (sqtau *  (-2.0)*values[i]);
}

/**********************************************************************
 *     Update all values along line a specified number of times
 *********************************************************************/
__global__ void update(float *values_out, int tpoints, int nsteps)
{
   int i;
   int j = 1+threadIdx.x;
   int idx = j+blockIdx.x*Num;

   if(idx<=tpoints){
       float values;
       float newval;
       float oldval;
       float x, fac, tmp;
       fac = 2.0*PI;
       tmp = tpoints-1;
       x=(float)(idx-1)/tmp;
       values = sin(fac*x);
       oldval = values;
       /* Update values for each time step */
       for (i = 1; i<= nsteps; i++) {
          if((idx==1) || (idx==tpoints))
              newval = 0.0;
          else
	      newval = (2.0*values)-oldval+(0.09*(-2.0*values));
          oldval = values;
	  values = newval;
       }
       values_out[idx] = values;
    }
}

/**********************************************************************
 *     Print final results
 *********************************************************************/
void printfinal()
{
   int i;

   for (i = 1; i <= tpoints; i++) {
      printf("%6.4f ", values[i]);
      if (i%10 == 0)
         printf("\n");
   }
}

/**********************************************************************
 *	Main program
 *********************************************************************/
int main(int argc, char *argv[])
{
	sscanf(argv[1],"%d",&tpoints);
	sscanf(argv[2],"%d",&nsteps);
        int size = (1+tpoints)*sizeof(float);
	check_param();
        float *values_dev;
        hipMalloc((void**)&values_dev, size);
	printf("Initializing points on the line...\n");
	//init_line();
	printf("Updating all points for all time steps...\n");
	if(tpoints%Num){
            update<<<1+tpoints/Num, Num>>>(values_dev, tpoints, nsteps);
        }else{
            update<<<tpoints/Num, Num>>>(values_dev, tpoints, nsteps);
        }

        hipMemcpy(values, values_dev, size, hipMemcpyDeviceToHost);
	hipFree(values_dev);

	printf("Printing final results...\n");
	printfinal();
	printf("\nDone.\n\n");
	return 0;
}
