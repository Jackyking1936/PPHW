
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 512

__global__ void add(int *a, int *b, int *c){
	c[blockIdx.x] = a[blockIdx.x]+b[blockIdx.x];
}

void random_ints(int *a, int n){
	for(int i=0; i<n; i++){
		a[i] = rand()%10+1;
	}
}

int main(void){

	int *a,*b,*c;
	int *d_a,*d_b,*d_c;
	int size = N*sizeof(int);

	hipMalloc((void**)&d_a, size);
	hipMalloc((void**)&d_b, size);
	hipMalloc((void**)&d_c, size);

	a=(int*)malloc(size); random_ints(a, N);
	b=(int*)malloc(size); random_ints(b, N);
	c=(int*)malloc(size);

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	add<<<N,1>>>(d_a, d_b, d_c);

	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	
	for(int i=0; i<N; i++)
		printf("Hello World!%d %d %d\n", a[i], b[i], c[i]);
	
	free(a); free(b);free(c);
	return 0;
}
